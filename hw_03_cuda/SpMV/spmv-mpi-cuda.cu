#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cmdline.h"
#include "input.h"
#include "config.h"
#include "timer.h"
#include "formats.h"

#define max(a,b) \
({ __typeof__ (a) _a = (a); \
   __typeof__ (b) _b = (b); \
 _a > _b ? _a : _b; })

#define min(a,b) \
({ __typeof__ (a) _a = (a); \
   __typeof__ (b) _b = (b); \
 _a < _b ? _a : _b; })

void usage(int argc, char** argv)
{
    printf("Usage: %s [my_matrix.mtx]\n", argv[0]);
    printf("Note: my_matrix.mtx must be real-valued sparse matrix in the MatrixMarket file format.\n"); 
}


void coo_spmv(coo_matrix* coo, const float* x, float* y) {
	for (int i = 0; i < coo->num_nonzeros; i++){   
		y[coo->rows[i]] += coo->vals[i] * x[coo->cols[i]];
	}
}

__global__ void coo_spmv_kernel(int num_nonzeros, const int* rows, const int* cols, 
								const float* vals, const float* x, float* y) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_nonzeros) {
		atomicAdd(&y[rows[idx]], vals[idx] * x[cols[idx]]);
	}
}

void coo_spmv_cuda(coo_matrix* coo, const float* x, float* y) {
    // Device memory pointers
    int *d_rows, *d_cols;
    float *d_vals, *d_x, *d_y;
    
    // Allocate device memory
    hipMalloc(&d_rows, coo->num_nonzeros * sizeof(int));
    hipMalloc(&d_cols, coo->num_nonzeros * sizeof(int));
    hipMalloc(&d_vals, coo->num_nonzeros * sizeof(float));
    hipMalloc(&d_x, coo->num_cols * sizeof(float));
    hipMalloc(&d_y, coo->num_rows * sizeof(float));
    
    // Copy data to device
    hipMemcpy(d_rows, coo->rows, coo->num_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, coo->cols, coo->num_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vals, coo->vals, coo->num_nonzeros * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, coo->num_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, coo->num_rows * sizeof(float));  // Initialize output to zero
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (coo->num_nonzeros + threadsPerBlock - 1) / threadsPerBlock;
    
    coo_spmv_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        coo->num_nonzeros, d_rows, d_cols, d_vals, d_x, d_y
    );
    
    // Copy result back to host
    hipMemcpy(y, d_y, coo->num_rows * sizeof(float), hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_vals);
    hipFree(d_x);
    hipFree(d_y);
}


// MIN_ITER, MAX_ITER, TIME_LIMIT, 
double benchmark_coo_spmv(coo_matrix * coo, float* x, float* y)
{
    int num_nonzeros = coo->num_nonzeros;

    // warmup    
    timer time_one_iteration;
    timer_start(&time_one_iteration);
	coo_spmv_cuda(coo, x, y);
    double estimated_time = seconds_elapsed(&time_one_iteration); 
//    printf("estimated time for once %f\n", (float) estimated_time);

    // determine # of seconds dynamically
    int num_iterations;
    num_iterations = MAX_ITER;

    if (estimated_time == 0)
        num_iterations = MAX_ITER;
    else {
        num_iterations = min(MAX_ITER, max(MIN_ITER, (int) (TIME_LIMIT / estimated_time)) ); 
    }
    printf("\tPerforming %d iterations\n", num_iterations);
    // time several SpMV iterations
    timer t;
    timer_start(&t);
    for(int j = 0; j < num_iterations; j++)
        // for (int i = 0; i < num_nonzeros; i++){   
        //     y[coo->rows[i]] += coo->vals[i] * x[coo->cols[i]];
        // }
		coo_spmv_cuda(coo, x, y);
    double msec_per_iteration = milliseconds_elapsed(&t) / (double) num_iterations;
    double sec_per_iteration = msec_per_iteration / 1000.0;
    double GFLOPs = (sec_per_iteration == 0) ? 0 : (2.0 * (double) coo->num_nonzeros / sec_per_iteration) / 1e9;
    double GBYTEs = (sec_per_iteration == 0) ? 0 : ((double) bytes_per_coo_spmv(coo) / sec_per_iteration) / 1e9;
    printf("\tbenchmarking COO-SpMV: %8.4f ms ( %5.2f GFLOP/s %5.1f GB/s)\n", msec_per_iteration, GFLOPs, GBYTEs); 

    return msec_per_iteration;
}

int main(int argc, char** argv)
{
    if (get_arg(argc, argv, "help") != NULL){
        usage(argc, argv);
        return 0;
    }

    char * mm_filename = NULL;
    if (argc == 1) {
        printf("Give a MatrixMarket file.\n");
        return -1;
    } else 
        mm_filename = argv[1];

    coo_matrix coo;
    read_coo_matrix(&coo, mm_filename);

    // fill matrix with random values: some matrices have extreme values, 
    // which makes correctness testing difficult, especially in single precision
    srand(13);
    for(int i = 0; i < coo.num_nonzeros; i++) {
        coo.vals[i] = 1.0 - 2.0 * (rand() / (RAND_MAX + 1.0)); 
	}
    
    printf("\nfile=%s rows=%d cols=%d nonzeros=%d\n", mm_filename, coo.num_rows, coo.num_cols, coo.num_nonzeros);
    fflush(stdout);

#ifdef TESTING
//print in COO format
    printf("Writing matrix in COO format to test_COO ...");
    FILE *fp = fopen("test_COO", "w");
    fprintf(fp, "%d\t%d\t%d\n", coo.num_rows, coo.num_cols, coo.num_nonzeros);
    fprintf(fp, "coo.rows:\n");
    for (int i=0; i<coo.num_nonzeros; i++)
    {
      fprintf(fp, "%d  ", coo.rows[i]);
    }
    fprintf(fp, "\n\n");
    fprintf(fp, "coo.cols:\n");
    for (int i=0; i<coo.num_nonzeros; i++)
    {
      fprintf(fp, "%d  ", coo.cols[i]);
    }
    fprintf(fp, "\n\n");
    fprintf(fp, "coo.vals:\n");
    for (int i=0; i<coo.num_nonzeros; i++)
    {
      fprintf(fp, "%f  ", coo.vals[i]);
    }
    fprintf(fp, "\n");
    fclose(fp);
    printf("... done!\n");
#endif 

    //initialize host arrays
    float * x = (float*)malloc(coo.num_cols * sizeof(float));
	float * y = (float*)malloc(coo.num_rows * sizeof(float));

    for(int i = 0; i < coo.num_cols; i++) {
        x[i] = rand() / (RAND_MAX + 1.0); 
    }
    for(int i = 0; i < coo.num_rows; i++)
        y[i] = 0;

    /* Benchmarking */
    double coo_gflops;
    coo_gflops = benchmark_coo_spmv(&coo, x, y);

    /* Test correctnesss */
	#ifdef TESTING
	float * y_exp = (float*)malloc(coo.num_rows * sizeof(float));
	coo_spmv(&coo, x, y_exp);
	float max_diff = 0;
	for(int i = 0; i < coo.num_rows; i++) {
		max_diff = max(max_diff, fabs(y[i] - y_exp[i]));
	}
	printf("Max difference: %f\n", max_diff);
	free(y_exp);

	#endif

    delete_coo_matrix(&coo);
    free(x);
    free(y);

    return 0;
}

